//*****************************************************************************************//
//  sobel_kernel.cu - CUDA Hough Transform Benchmark
//
//  Authors: Ramnarayan Krishnamurthy, University of Colorado (Shreyas.Ramnarayan@gmail.com)
//	         Matthew Demi Vis, Embry-Riddle Aeronautical University (MatthewVis@gmail.com)
//			 
//	This code was used to obtain results documented in the SPIE Sensor and Technologies paper: 
//	S. Siewert, V. Angoth, R. Krishnamurthy, K. Mani, K. Mock, S. B. Singh, S. Srivistava, 
//	C. Wagner, R. Claus, M. Demi Vis, “Software Defined Multi-Spectral Imaging for Arctic 
//	Sensor Networks”, SPIE Algorithms and Technologies for Multipectral, Hyperspectral, and 
//	Ultraspectral Imagery XXII, Baltimore, Maryland, April 2016. 
//
//	This code was developed for, tested and run on a Jetson TK1 development kit by NVIDIA
//  running Ubuntu 14.04 
//	
//	Please use at your own risk. We are sharing so that other researchers and developers can 
//	recreate our results and make suggestions to improve and extend the benchmarks over time.
//
//*****************************************************************************************//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define MAXRGB	 	255

//***************************************************************//
// Sobel transform using CUDA hardware
//***************************************************************//
__global__ void CUDA_transform(unsigned char *img_out, unsigned char *img_in, unsigned int width, unsigned int height){
	int x,y;
	unsigned char LUp,LCnt,LDw,RUp,RCnt,RDw;
	int pixel;
	
	x=blockDim.x*blockIdx.x+threadIdx.x;
	y=blockDim.y*blockIdx.y+threadIdx.y;
	
	if( x<width && y<height )
	{
		LUp = (x-1>=0 && y-1>=0) ? img_in[(x-1)+(y-1)*width] : 0;
		LCnt= (x-1>=0)           ? img_in[(x-1)+y*width]:0;
		LDw = (x-1>=0 && y+1<height) ? img_in[(x-1)+(y+1)*width] : 0;
		RUp = (x+1<width && y-1>=0)  ? img_in[(x+1)+(y-1)*width] : 0;
		RCnt= (x+1<width)            ? img_in[(x+1)+y*width] : 0;
		RDw = (x+1<width && y+1<height) ? img_in[(x+1)+(y+1)*width] : 0;
		pixel = -1*LUp  + 1*RUp +
		-2*LCnt + 2*RCnt +
		-1*LDw  + 1*RDw;
		pixel = (pixel<0) ? 0 : pixel;
		pixel = (pixel>MAXRGB) ? MAXRGB : pixel;
		img_out[x+y*width] = pixel;
	}
}

//***************************************************************//
// Sobel transform using the CPU
//***************************************************************//
void CPU_transform(unsigned char *img_out, unsigned char *img_in, unsigned int width, unsigned int height) {
	unsigned char LUp,LCnt,LDw,RUp,RCnt,RDw;
	int pixel;
	for(int y=0; y<height; y++)
	{
		for(int x=0; x<width; x++)
		{
			#ifdef DEBUG
				printf("Pixel X:%d Y:%d\n",x,y);
			#endif
			assert(x+(y*width)<width*height);
			LUp = (x-1>=0 && y-1>=0)? img_in[(x-1)+(y-1)*width]:0;
			LCnt= (x-1>=0)? img_in[(x-1)+y*width]:0;
			LDw = (x-1>=0 && y+1<height)? img_in[(x-1)+(y+1)*width]:0;
			RUp = (x+1<width && y-1>=0)? img_in[(x+1)+(y-1)*width]:0;
			RCnt= (x+1<width)? img_in[(x+1)+y*width]:0;
			RDw = (x+1<width && y+1<height)? img_in[(x+1)+(y+1)*width]:0;
			pixel = -1*LUp  + 1*RUp + -2*LCnt + 2*RCnt + -1*LDw  + 1*RDw;
			pixel=(pixel<0)?0:pixel;
			pixel=(pixel>MAXRGB)?MAXRGB:pixel;
			img_out[x+y*width]=pixel;
			#ifdef DEBUG
				printf("\r%5.2f",100*(float)(y*width+x)/(float)(width*height-1));            
			#endif
		}
	}
#ifdef DEBUG
	printf("\n");
#endif
}
